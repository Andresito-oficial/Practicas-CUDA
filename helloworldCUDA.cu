#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void hello_kernel()
{
    // device-side printf is supported in CUDA (requires proper compute capability)
    printf("Hello from GPU (block %d thread %d)\n", blockIdx.x, threadIdx.x);
}

int main()
{
    // launch one block with one thread
    hello_kernel<<<1, 1>>>();

    // wait for kernel to finish and check for errors
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;n
    }

    std::cout << "Hello from host!" << std::endl;
    return 0;
}